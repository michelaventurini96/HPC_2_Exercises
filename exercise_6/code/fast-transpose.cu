/*Ex. 5
1. implement matrix transpose in cuda with shared memory
  (use block algorithm/FAST TRANSPOSE).
2. MATRIX SIZE = 8192X8192 and register the time of solution compared with
  naive one for 64,512, 1024 threads/block.
3. Try to reach mem band = 100 Gb/s.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
// kernels transpose a tile of TILE_DIM x TILE_DIM elements
// using a TILE_DIM x BLOCK_ROWS thread block, so that each thread
// transposes TILE_DIM/BLOCK_ROWS elements. TILE_DIM is an
// integral multiple of BLOCK_ROWS
#define TILE_DIM 32
//#define BLOCK_ROWS 2 //64 threads/block
//#define BLOCK_ROWS 16 //512 threads/block
#define BLOCK_ROWS 32 //1024 threads/block
// Number of repetitions used for timing.
#define NUM_REPS 100

__host__ void printMatrix(const float* const data, const size_t size_x, const size_t size_y, const size_t a, const size_t b){
  for(size_t i=0; i<a; i++){
  	printf("%5.2f ", data[i]);
  }
  printf("\n");
}

__host__ int compareRes(const float* const odata, const float* const gold, const size_t msize){
  int res = 0;
  for(size_t i = 0; i<msize; i++){
    if (odata[i] != gold[i]) ++res;
  }
  return res;
}

__global__ void transposeNaive(float *odata, const float* const idata,
    const int width, const int height) {
  int xIndex = blockIdx.x*TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y*TILE_DIM + threadIdx.y;
  int index_in = xIndex + width * yIndex;
  int index_out = yIndex + height * xIndex;

    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
        odata[index_out+i] = idata[index_in+i*width];
    }
}

__host__ void computeTransposeHost(float* odata, const float* const h_idata,
    const int size_x, const int size_y){
  for (size_t i = 0; i < size_x; i++) {
    for (size_t j = 0; j < size_y; j++) {
      odata[j+i*size_x] = h_idata[i + j*size_y];
    }
  }
}

__global__ void transposeCoalesced(float *odata, const float *const idata, const int width,
    const int height){
  __shared__ float tile[TILE_DIM][TILE_DIM+1]; //padding so different threads access different shared memory banks and no bottleneck

  int xIndex = blockIdx.x*TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y*TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) { // use a tile like a buffer to transpose blocks
      tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
  }

  __syncthreads();
  for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
  }
}

int main( int argc, char** argv) {
  
// set matrix size
  const int size_x = 8192;
  const int size_y = 8192;

// execution configuration parameters
  dim3 grid(size_x/TILE_DIM, size_y/TILE_DIM), threads(TILE_DIM,BLOCK_ROWS);
  
// CUDA events
  hipEvent_t start, stop;
  
// size of memory required to store the matrix
  const int mem_size = sizeof(float) * size_x*size_y;
  
// allocate host memory
  float *h_idata = (float*) malloc(mem_size);
  float *h_odata = (float*) malloc(mem_size);
  float *transposeHost = (float *) malloc(mem_size);
  
// allocate device memory
  float *d_idata, *d_odata;
  hipMalloc( (void**) &d_idata, mem_size);
  hipMalloc( (void**) &d_odata, mem_size);
  
// initalize host data
  for(int i = 0; i < (size_x*size_y); ++i) 
	h_idata[i] = (float) i;
  printf("Initial matrix:\n");
  printMatrix(h_idata, size_x, size_y, 10, 10);
 
// copy host data to device
  hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice );

// Compute reference transpose solution
  computeTransposeHost(transposeHost, h_idata, size_x, size_y);
  printf("Reference solution:\n");
  printMatrix(transposeHost, size_x, size_y, 10, 10);


  printf("\nMatrix size: %dx%d, tile: %dx%d, block: %dx%d\n\n",
  	size_x, size_y, TILE_DIM, TILE_DIM, TILE_DIM, BLOCK_ROWS);
  printf("Kernel\t\t\tLoop over kernel\n");
  printf("------\t\t\t----------------\n");

  
// initialize events, EC parameters
  hipEventCreate(&start);
  hipEventCreate(&stop);

//*********************************time transposeNaive*************************************
  hipEventRecord(start, 0);
  for (int i=0; i < NUM_REPS; i++) {
    transposeNaive<<<grid, threads>>>(d_odata, d_idata,size_x,size_y);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float outerTime;
  hipEventElapsedTime(&outerTime, start, stop);

  hipMemcpy(h_odata,d_odata, mem_size, hipMemcpyDeviceToHost);
  
//check correctness of solution
  int res = compareRes(transposeHost, h_odata, size_x*size_y);
  if (res != 0) printf("*** transposeNaive kernel FAILED ***\n");
  else printf("***transposeNaive kernel OK***\n");
// report effective bandwidths
  float outerBandwidth = 2.*1000*mem_size/(1024*1024*1024)/(outerTime/NUM_REPS);
  
  printf("transposeNaive\t\t%5.2f GB/s\n", outerBandwidth);
  //printf("transposeNaive:\n");
  //printMatrix(h_odata, size_x, size_y, 10, 10);

//*********************************time transposeCoalesced************************************
  hipEventRecord(start, 0);
  for (int i=0; i < NUM_REPS; i++) {
      transposeCoalesced<<<grid, threads>>>(d_odata, d_idata,size_x,size_y);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  //float outerTime1;
  hipEventElapsedTime(&outerTime, start, stop);
  hipMemcpy(h_odata,d_odata, mem_size, hipMemcpyDeviceToHost);
  
//check corrctness of result
  res = compareRes(transposeHost, h_odata, size_x*size_y);
  if (res != 0) printf("*** transposeCoalasced kernel FAILED ***\n");
  else printf("***transposeCoalasced kernel OK***\n");
// report effective bandwidths
  outerBandwidth = 2.*1000*mem_size/(1024*1024*1024)/(outerTime/NUM_REPS);
    
  printf("transposeCoalesced\t%5.2f GB/s\n", outerBandwidth);
  //printf("transposeCoalesced\n");
  //printMatrix(h_odata, size_x, size_y, 10, 10);
  
// cleanup memory
  free(h_idata); free(h_odata); free(transposeHost);
  hipFree(d_idata); hipFree(d_odata);
  hipEventDestroy(start); hipEventDestroy(stop);

  return 0;
}
